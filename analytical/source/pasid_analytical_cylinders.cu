#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <fstream>
#include <stdio.h>
#include <random>
//This program simulates restricted diffusion with exchange in a substrate of regular cylinders defined analytically
//Cylinders may have any packing and any size distribution as long as they are parallel


//simulation options; will be read from an options txt file
struct options
{
	long long Npart;
	double T;
	double sim_dt;
	bool do_samp;
	double samp_dt;
	long long n_dim;
	double D0;
	long long sim_Nt;
	long long save_Nt;
	double ds;
	long long N_save; //N time points x N particles
	long long N_sim;
	bool save_states; //save particle state history to file or not
	double kappa; //membrane permeability
};


//world data
struct world
{
	long long num_cells, num_voxels;
	double max_x, max_y, max_z, x_length, y_length, z_length, f1, vox_size;
};

__global__ void random_init(hiprandState* states)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(index, index, 0, &states[index]);
}


__device__ void move(double& tmp_x, double& tmp_y, double& tmp_z, double& tmp_dx,
	double& tmp_dy, double& tmp_dz, int entry, hiprandState* states, int index, options* opt)
{
	tmp_dx = hiprand_normal(&states[index]);
	tmp_dy = hiprand_normal(&states[index]);
	tmp_dz = hiprand_normal(&states[index]);

	double norm = (*opt).ds * rnorm3d(tmp_dx, tmp_dy, tmp_dz);

	tmp_dx *= norm;
	tmp_dy *= norm;
	tmp_dz *= norm;

	tmp_x += tmp_dx;
	tmp_y += tmp_dy;
	tmp_z += tmp_dz;
}

__device__ void restrict_to_world(double& e_x, double& e_y, double& e_z, options* opt, world* w, double& tmp_x, double& tmp_y, double& tmp_z)
{
	if (tmp_x < -w->max_x) { tmp_x += w->x_length; e_x -= w->x_length; }
	if (tmp_x >= w->max_x) { tmp_x -= w->x_length; e_x += w->x_length; }

	if (tmp_y < -w->max_y) { tmp_y += w->y_length; e_y -= w->y_length; }
	if (tmp_y >= w->max_y) { tmp_y -= w->y_length; e_y += w->y_length; }

	if (tmp_z < -w->max_z) { tmp_z += w->z_length; e_z -= w->z_length; }
	if (tmp_z >= w->max_z) { tmp_z -= w->z_length; e_z += w->z_length; }
}


__device__ void pair(long long x, long long y, long long& xy)
{
	//device function for pairing particle coordinates
	x >= 0 ? x = 2 * x : x = -2 * x - 1;
	y >= 0 ? y = 2 * y : y = -2 * y - 1;

	x >= y ? xy = x * x + x + y : xy = y * y + x;
}


//binary search without recursion to avoid potentially filling the stack
__device__ long long binary_search_iter(long long* A, long long lower, long long upper, long long x)
{
	while (upper >= lower) {
		long long mid = lower + (upper - lower) / 2;
		if (A[mid] == x) return mid;
		(A[mid] > x) ? upper = mid - 1 : lower = mid + 1;
	}
	return -1;
}



__device__ long long is_particle_in_any_cell(double tmp_x, double tmp_y, world* w, double* centre_x, double* centre_y, double* radii, long long* table, long long* cell_idx)
{
	long long x_pos = floor(tmp_x / w->vox_size);
	long long y_pos = floor(tmp_y / w->vox_size);
	long long xy; //voxel identifier
	long long which_cell, which_voxel;
	long long inside = 0; //zero means outside all cells
	double distance, r, cx, cy;
	
	pair(x_pos, y_pos, xy); //get the identifier


	which_voxel = binary_search_iter(table, 0, w->num_voxels - 1, xy); //iterative binary search	
	
	//printf("--------\n");
	//printf("which_voxel is: %lld\n", which_voxel);
	//printf("xy is: %lld\n", xy);
	
	which_cell = cell_idx[which_voxel]-1; //index of cell containing the voxel containing the particle
	//note the minus 1 to take into account that MATLAB numbering starts at 1. Need to fix this later
				
    //printf("which_cell is: %lld\n", which_cell);
    
    r = radii[which_cell];
    cx = centre_x[which_cell];
    cy = centre_y[which_cell];

	//printf("cell_idx is: %lld\n", which_cell);
	
	if (which_cell >= 0) //-1 means voxel is not in any cell
	{
		distance = (tmp_x-cx)*(tmp_x-cx) + (tmp_y-cy)*(tmp_y-cy);
		if ( distance <= (r*r)  ) //means particle is in this cell
		{
			inside = 1;
		}
	}
	/*/
	printf("Is inside? : %lld\n", inside);
	printf("tmp_x : %f\n", tmp_x*1e6);
	printf("tmp_y : %f\n", tmp_y*1e6);
	printf("Dist : %f\n", distance*1e12);
		printf("cx : %f\n", cx*1e6);
		printf("cy : %f\n", cy*1e6);
		printf("r : %f\n", r*1e6);
	/*/
	
	return inside;
}

__device__ void check_state(world* w, options* opt, double& tmp_x, double& tmp_y, double& tmp_z, double& tmp_dx,
	double& tmp_dy, double& tmp_dz, long long& tmp_loc, double* centre_x, double* centre_y, double *radii, long long* table, long long * cell_idx,  hiprandState* states, int index)
{
	bool reject = false; //reject move or not
	double p_ex, p_12, p_21;
	long long inside;

	inside = is_particle_in_any_cell(tmp_x, tmp_y, w, centre_x, centre_y, radii, table, cell_idx); //determine whether particle is in any cell

	if (inside == 1) //"now intra"
	{
		if (tmp_loc == 1) { reject = false; } //was intra before
		else //was not intra before
		{
			//compute permeation probability p_21
			p_ex = (double)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0)); //from Szafer 1995 and others, for any geoemetry
			p_21 = (double)p_ex * w->f1;
			if (hiprand_uniform_double(&states[index]) < p_21) { reject = false; tmp_loc = 1; }
			else { reject = true; tmp_loc = 0; }
		}
	}

	if (inside == 0) //"now extra"
	{
		if (tmp_loc == 0) { reject = false; } //was extra before
		else //was not extra before
		{
			//compute permeation probability p_12
			p_ex = (double)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0)); //from Szafer 1995 and others, for any geoemetry
			p_12 = (double)p_ex * (1 - w->f1);
			if (hiprand_uniform(&states[index]) < p_12) { reject = false; tmp_loc = 0; }
			else { reject = true; tmp_loc = 1; }
		}
	}


	if (reject) { tmp_x -= tmp_dx; tmp_y -= tmp_dy; tmp_z -= tmp_dz; }

}

__global__ //the global keyword tells compiler this is device code not host code
void engine(double* x, double* y, double* z, long long* loc, double* centre_x, double* centre_y, double* radii, long long* table, long long* cell_idx,
	hiprandState* states, options* opt, world* w)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;

	long entry;
	long save;
	long save_c_t;
	double tmp_x, tmp_y, tmp_z;
	double tmp_dx, tmp_dy, tmp_dz;
	double e_x, e_y, e_z; //keep track of hyperposition
	long long tmp_loc;


	//for (int c_p = index; c_p < (*opt).Npart; c_p += stride)
	long c_p = index;
	if (c_p < opt->Npart) //ensure we keep within bounds
	{
		tmp_x = x[(long)(c_p * (*opt).save_Nt)];
		tmp_y = y[(long)(c_p * (*opt).save_Nt)];
		tmp_z = z[(long)(c_p * (*opt).save_Nt)];
		tmp_dx = 0;
		tmp_dy = 0;
		tmp_dz = 0;
		tmp_loc = loc[(long)(c_p * (*opt).save_Nt)];

		e_x = 0; e_y = 0; e_z = 0;
		//adding a delay loop here
		/*
		for (int c_t = 0; c_t < (*opt).sim_Nt; c_t++)
		{
			move(tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, entry, states, index, opt);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
			check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, tmp_loc, table, mask, states, index, vox_idx);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
		}

		e_x = 0; e_y = 0; e_z = 0;*/

		save_c_t = -1;
		save = 0;

		for (int c_t = 0; c_t < (*opt).sim_Nt; c_t++)
		{

			move(tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, entry, states, index, opt);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
			check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, tmp_loc, centre_x, centre_y, radii, table, cell_idx, states, index);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);

			save++;

			if (save == (long)(opt->samp_dt / opt->sim_dt))
			{
				save_c_t++;
				entry = c_p * (*opt).save_Nt + save_c_t;
				x[entry] = tmp_x + e_x;
				y[entry] = tmp_y + e_y;
				z[entry] = tmp_z + e_z;
				loc[entry] = tmp_loc;
				save = 0;
			}
		}
	}
}



void set_options(options* opt, char* r_fn, char* s_fn, char* g_fn)
{
	//sets options loaded from file
	std::string opt_fn = "C:\\Users\\Arthur\\source\\repos\\PaSiD\\pasid_analytical_cylinders_opt.txt";
	std::string dummy;

	std::ifstream tf;
	tf.open(opt_fn, std::ios::in);
	tf >> dummy >> opt->Npart;
	tf >> dummy >> opt->T;
	tf >> dummy >> opt->sim_dt;
	tf >> dummy >> opt->do_samp;
	tf >> dummy >> opt->samp_dt;
	tf >> dummy >> opt->n_dim;
	tf >> dummy >> opt->D0;
	tf >> dummy >> opt->kappa;
	tf >> dummy >> g_fn;
	tf >> dummy >> r_fn;
	tf >> dummy >> opt->save_states;
	tf >> dummy >> s_fn;
	tf.close();

	opt->sim_Nt = (long long)round(opt->T / opt->sim_dt);
	opt->save_Nt = (long long)round(opt->T / opt->samp_dt);
	opt->ds = (double)sqrt(2 * (*opt).n_dim * (*opt).D0 * (*opt).sim_dt); //step size
	opt->N_save = (long long)opt->Npart * opt->save_Nt; //N time points x N particles
	opt->N_sim = (long long)opt->Npart * opt->sim_Nt;

	std::cout << "Loaded options from: " << opt_fn << std::endl;
}


void save_trajectory(double* x, double* y, double* z, char* r_fn, options* opt)
{
	std::cout << "Saving trajectory to: " << r_fn << std::endl;
	FILE* tf;
	tf = fopen(r_fn, "wb");
	fwrite(&(opt->Npart), sizeof(long long), 1, tf);
	fwrite(&(opt->T), sizeof(double), 1, tf);
	fwrite(&(opt->save_Nt), sizeof(long long), 1, tf);
	fwrite(x, sizeof(double), opt->N_save, tf); // opt->N_save
	fwrite(y, sizeof(double), opt->N_save, tf);
	fwrite(z, sizeof(double), opt->N_save, tf);
	fclose(tf);
	std::cout << "Done." << std::endl;
}


void save_state_history(long long* s, char* s_fn, options* opt)
{
	//saves history of particle identities/compartment identities
	std::cout << "Saving state history to: " << s_fn << std::endl;
	FILE* tf;
	tf = fopen(s_fn, "wb");
	fwrite(&(opt->Npart), sizeof(long long), 1, tf);
	fwrite(&(opt->T), sizeof(double), 1, tf);
	fwrite(&(opt->save_Nt), sizeof(long long), 1, tf);
	fwrite(s, sizeof(long long), opt->N_save, tf); // opt->N_save
	fclose(tf);
	std::cout << "Done." << std::endl;
}


void get_num_cells(world* w, char* g_fn)
{
	//open substrate file and get number of cells and number of voxels
	FILE* sf;
	sf = fopen(g_fn, "rb");
	fread(&(w->num_cells), sizeof(long long), 1, sf);
	fread(&(w->num_voxels), sizeof(long long), 1, sf);
	fclose(sf);
}


void load_substrate(double* h_centre_x, double* h_centre_y, double* h_radii, long long* h_table, long long *h_cell_idx, world* w, char* g_fn)
{
	//load substrate from file
	FILE* sf;
	sf = fopen(g_fn, "rb");
	fread(&(w->num_cells), sizeof(double), 1, sf);
	fread(&(w->num_voxels), sizeof(double), 1, sf);
	fread(&(w->max_x), sizeof(double), 1, sf);
	fread(&(w->max_y), sizeof(double), 1, sf);
	fread(&(w->max_z), sizeof(double), 1, sf);
	fread(&(w->vox_size), sizeof(double), 1, sf);
	fread(&w->f1, sizeof(double), 1, sf);
	fread(h_centre_x, sizeof(double), w->num_cells, sf); 
	fread(h_centre_y, sizeof(double), w->num_cells, sf); 
	fread(h_radii, sizeof(double), w->num_cells, sf); 
	fread(h_table, sizeof(long long), w->num_voxels, sf); 
	fread(h_cell_idx, sizeof(long long), w->num_voxels, sf); 
	fclose(sf);

	w->x_length = 2 * w->max_x;
	w->y_length = 2 * w->max_y;
	w->z_length = 2 * w->max_z;

	std::cout << "Loaded substrate from: " << g_fn << std::endl;
}


__global__ void generate_initial_distribution(double* x, double* y, double* z, long long* loc, double* centre_x,
	double* centre_y, double* radii, long long* table, long long* cell_idx, hiprandState* states, options* opt, world* w)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; //get thread idx
	double tmp_x, tmp_y, tmp_z, frac = 0.5; //frac defines intra-extra split of initial populations
	long long idx, inside;
	bool success_1 = false, success_2 = false, success = false;
	if (w->f1 == 0) { success_2 = true; frac = 1; }
	int N1 = (int)(frac * opt->Npart);
	if (index < N1)
	{
		//places particles in initial positions all over substrate

		while (!success)
		{
			tmp_x = -w->max_x + 2 * hiprand_uniform_double(&states[index]) * w->max_x; //suggest initial position
			tmp_y = -w->max_y + 2 * hiprand_uniform_double(&states[index]) * w->max_y; //suggest initial position
			tmp_z = -w->max_z + 2 * hiprand_uniform_double(&states[index]) * w->max_z; //suggest initial position
			
			inside = is_particle_in_any_cell(tmp_x, tmp_y, w, centre_x, centre_y, radii, table, cell_idx); //determine whether particle is in any cell
			
			//if (index == 1 && inside == 1) {printf("cell_idx is: %lld\n", inside); printf("x_pos: %f\n", tmp_x*1e6);  printf("y_pos: %f\n", tmp_y*1e6);  printf("z_pos: %f\n", tmp_z*1e6);};
			//printf("cell_idx is: %lld\n", inside);
			
			if (inside == 1 && !success_1)
			{
				idx = index * opt->save_Nt;
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 1;
				success_1 = true;
			}


			if (inside == 0 && !success_2)
			{
				idx = (N1 + index) * opt->save_Nt;
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 0;
				success_2 = true;
			}
			success = success_1 && success_2;
		}
	}
}



int main(void)
{

	std::clock_t start;
	double duration;
	start = std::clock();

	hipError_t error = hipSuccess;

	int nDevices;
	hipGetDeviceCount(&nDevices);
	printf("Number of devices: %d\n", nDevices);
	int activeDevice;
	hipGetDevice(&activeDevice);
	printf("Active device index: %d\n", activeDevice);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, activeDevice);
	printf("Device name: %s\n", prop.name);


	//non-numerical options, host-only
	char r_fn[200], s_fn[200], g_fn[200]; //filenames for trajectories, states and substrate

	//world structure for device and host
	world* h_w, * dev_w;
	h_w = (world*)malloc(sizeof(world));
	hipMalloc(&dev_w, sizeof(world));


	//options structure for host
	options* opt, * dev_opt;
	opt = (options*)malloc(sizeof(options));
	set_options(opt, r_fn, s_fn, g_fn);
	hipMalloc(&dev_opt, sizeof(options));
	hipMemcpy(dev_opt, opt, sizeof(options), hipMemcpyHostToDevice);

	//we will load the simulation world from file, no need to waste time implementing it in here
	//the first entry in the substrate file will be the number of cells in the world
	//this is so we know how large the world arrays centre_x, centre_y, radii need to be

	get_num_cells(h_w, g_fn);
	std::cout << "Num cells: " << h_w->num_cells << "Num voxels: " << h_w->num_voxels << std::endl;
	//now we declare world arrays on device and host
	double* h_centre_x, *h_centre_y, *h_radii;
	double* dev_centre_x, *dev_centre_y, *dev_radii;
	long long *h_table, *h_cell_idx;
	long long *dev_table, *dev_cell_idx;
	
	
	//allocate on host
	h_centre_x = (double*)malloc(h_w->num_cells * sizeof(double));
	h_centre_y = (double*)malloc(h_w->num_cells * sizeof(double));
	h_radii = (double*)malloc(h_w->num_cells * sizeof(double));
	h_table = (long long*)malloc(h_w->num_voxels * sizeof(long long));
	h_cell_idx = (long long*)malloc(h_w->num_voxels * sizeof(long long));
	
	//allocate on device
	hipMalloc(&dev_centre_x, h_w->num_cells * sizeof(double));
	hipMalloc(&dev_centre_y, h_w->num_cells * sizeof(double));
	hipMalloc(&dev_radii, h_w->num_cells * sizeof(double));
	hipMalloc(&dev_table, h_w->num_voxels * sizeof(long long));
	hipMalloc(&dev_cell_idx, h_w->num_voxels * sizeof(long long));
	
	//load the substrate
	load_substrate(h_centre_x, h_centre_y, h_radii, h_table, h_cell_idx, h_w, g_fn);

	std::cout << "Num cells: " << h_w->num_cells << " Diameter[0]: " << h_radii[0]*2 << 
	" max_z: " << h_w->max_z << " Num voxels: " << h_w->num_voxels  << 
	" Voxel size : " << h_w->vox_size << " Table(23) : " << h_table[22] << std::endl;

	//copy substrate data to GPU
	hipMemcpy(dev_w, h_w, sizeof(world), hipMemcpyHostToDevice);
	hipMemcpy(dev_centre_x, h_centre_x, h_w->num_cells * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_centre_y, h_centre_y, h_w->num_cells * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_radii, h_radii, h_w->num_cells * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_table, h_table, h_w->num_voxels * sizeof(long long), hipMemcpyHostToDevice);
	error = hipMemcpy(dev_cell_idx, h_cell_idx, h_w->num_voxels * sizeof(long long), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cout << "FAILED TO ALLOCATE SUBSTRATE MEMORY ON GPU." << std::endl;
		//throw error;
	}

	//declare traj arrays and particle location (compartment id)
	long long* h_loc, * dev_loc;
	double* h_x, * h_y, * h_z; //for the host
	double* dev_x, * dev_y, * dev_z; //for the device
	//allocate them on host
	h_loc = (long long*)malloc(opt->N_save * sizeof(long long));
	h_x = (double*)malloc(opt->N_save * sizeof(double));
	h_y = (double*)malloc(opt->N_save * sizeof(double));
	h_z = (double*)malloc(opt->N_save * sizeof(double));


	//allocate memory for arrays on device
	hipMalloc(&dev_loc, opt->N_save * sizeof(long long));
	hipMalloc(&dev_x, opt->N_save * sizeof(double));
	hipMalloc(&dev_y, opt->N_save * sizeof(double));
	hipMalloc(&dev_z, opt->N_save * sizeof(double));

	//copy x,y,z  and id arrays to device
	hipMemcpy(dev_loc, h_loc, opt->N_save * sizeof(long long), hipMemcpyHostToDevice);
	hipMemcpy(dev_x, h_x, opt->N_save * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, h_y, opt->N_save * sizeof(double), hipMemcpyHostToDevice);
	error = hipMemcpy(dev_z, h_z, opt->N_save * sizeof(double), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cout << "FAILED TO ALLOCATE TRAJECTORY MEMORY ON GPU." << std::endl;
		//throw error;
	}
	else { std::cout << "Generating initial distribution..." << std::endl; }

	//define grid texture
	int blockSize;
	512 > opt->Npart ? blockSize = (int)opt->Npart : blockSize = 512;
	int numBlocks = (int)(opt->Npart + blockSize - 1) / blockSize; //make sure to round up in case N is not an integer multiple of blockSize

	//allcoate hiprandState for every CUDA thread on the host
	hiprandState* dev_states;
	hipMalloc(&dev_states, blockSize * numBlocks * sizeof(hiprandState));
	//initialise RNG for all threads
	random_init << < numBlocks, blockSize >> > (dev_states);
	//generate initial particle distribution
	
	
	generate_initial_distribution << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_centre_x, dev_centre_y, dev_radii, dev_table, dev_cell_idx, dev_states, dev_opt, dev_w);


	std::cout << "Running simulation..." << std::endl;
	//launch simulation engine
	engine << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_centre_x, dev_centre_y, dev_radii, dev_table, dev_cell_idx, dev_states, dev_opt, dev_w);

	hipDeviceSynchronize(); //Tell CPU to wait until kernel is done before accessing results. This is necessary because
							//cuda kernel launches do not block the calling CPU thread.

	//std::cout << "Finished generating initial distribution..." << std::endl;

	//copy simulated trajectories back to host machine
	hipMemcpy(h_x, dev_x, opt->N_save * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_y, dev_y, opt->N_save * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_z, dev_z, opt->N_save * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_loc, dev_loc, opt->N_save * sizeof(long long), hipMemcpyDeviceToHost);
	
    //std::cout << "Example dev_loc(60): " << dev_loc[0] << std::endl;
	
	std::cout << "Saving results..." << std::endl;
	
	//write results to binary files
	save_trajectory(h_x, h_y, h_z, r_fn, opt);
	save_state_history(h_loc, s_fn, opt);

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Elapsed time is: " << duration << " seconds." << std::endl;


	// Free memory on host
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_w);
	free(opt);
	free(h_loc);
	free(h_centre_x);
	free(h_centre_y);
	free(h_radii);
	free(h_table);
	free(h_cell_idx);
	//free memory on device
	//free memory on device
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(dev_states);
	hipFree(dev_loc);
	hipFree(dev_w);
	hipFree(dev_opt);
	hipFree(dev_centre_x);
	hipFree(dev_centre_y);
	hipFree(dev_radii);
	hipFree(dev_table);
	hipFree(dev_cell_idx);
	return 0;
}
