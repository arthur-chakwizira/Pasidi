#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <string>
#include <fstream>
#include <stdio.h>
#include <random>
//This program simulates restricted diffusion with exchange in a substrate of anything
//Does not save trajectories
//Only saves signal

//simulation options; will be read from an options txt file
struct options
{
	long long Npart;
	double T;
	double sim_dt;
	double samp_dt;
	long long n_dim;
	double D0;
	long long sim_Nt;
	long long save_Nt;
	double ds;
	long long N_save; //N time points x N particles
	long long N_sim;
	bool save_states; //save particle state history to file or not
	double kappa; //membrane permeability
		//additional gwf options
	long long n_acq; //# b-values,
	long long n_gwf_points; // # time points in each waveform
	double max_b_value;
	double gamma; //gyromagnetic ratio
	long long delay; //number of time steps to take before acquiring signals
	bool save_positions; //save final positions or not; useful for debugging
};


//world data
struct world
{
	long long num_voxels;
	double vox_size, max_x, max_y, max_z, x_length, y_length, z_length, f1;
};

__global__ void random_init(hiprandState* states)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(clock64()+index, index, 0, &states[index]);
}


__device__ void pair(long long x, long long y, long long z, long long& xyz)
{
	//device function for pairing particle coordinates
	long long xy;
	x >= 0 ? x = 2 * x : x = -2 * x - 1;
	y >= 0 ? y = 2 * y : y = -2 * y - 1;
	z >= 0 ? z = 2 * z : z = -2 * z - 1;

	x >= y ? xy = x * x + x + y : xy = y * y + x;
	xy >= 0 ? xy = 2 * xy : xy = -2 * xy - 1;

	xy >= z ? xyz = xy * xy + xy + z : xyz = z * z + xy;
}


__device__ void find(long long* A, long long sizeA, long long a, long long& vox_idx)
{
	//find index of element in array
	for (int c = 0; c < sizeA; c++)
	{
		if (A[c] == a) { vox_idx = c; break; }
	}
}

//we will try binarySearch here instead of the slow find above
__device__ long long binary_search_rec(long long* A, long long lower, long long upper, long long x)
{
	if (upper >= lower) {
		long long mid = lower + (upper - lower) / 2;
		if (A[mid] == x)
			return mid;
		if (A[mid] > x)
			return binary_search_rec(A, lower, mid - 1, x);
		return binary_search_rec(A, mid + 1, upper, x);
	}
	return -1;
}

//binary search without recursion to avoid potentially filling the stack
__device__ long long binary_search_iter(long long* A, long long lower, long long upper, long long x)
{
	while (upper >= lower) {
		long long mid = lower + (upper - lower) / 2;
		if (A[mid] == x) return mid;
		(A[mid] > x) ? upper = mid - 1 : lower = mid + 1;
	}
	return -1;
}


__device__ void move(double& tmp_x, double& tmp_y, double& tmp_z, double& tmp_dx,
	double& tmp_dy, double& tmp_dz, int entry, hiprandState* states, int index, options* opt)
{
	tmp_dx = hiprand_normal(&states[index]);
	tmp_dy = hiprand_normal(&states[index]);
	tmp_dz = hiprand_normal(&states[index]);

	double norm = (*opt).ds * rnorm3d(tmp_dx, tmp_dy, tmp_dz);

	tmp_dx *= norm;
	tmp_dy *= norm;
	tmp_dz *= norm;

	tmp_x += tmp_dx;
	tmp_y += tmp_dy;
	tmp_z += tmp_dz;
}

__device__ void restrict_to_world(double& e_x, double& e_y, double& e_z, options* opt, world* w, double& tmp_x, double& tmp_y, double& tmp_z)
{
	if (tmp_x < -w->max_x) { tmp_x += w->x_length; e_x -= w->x_length; }
	if (tmp_x >= w->max_x) { tmp_x -= w->x_length; e_x += w->x_length; }

	if (tmp_y < -w->max_y) { tmp_y += w->y_length; e_y -= w->y_length; }
	if (tmp_y >= w->max_y) { tmp_y -= w->y_length; e_y += w->y_length; }

	if (tmp_z < -w->max_z) { tmp_z += w->z_length; e_z -= w->z_length; }
	if (tmp_z >= w->max_z) { tmp_z -= w->z_length; e_z += w->z_length; }
}


__device__ void check_state(world* w, options* opt, double& tmp_x, double& tmp_y, double& tmp_z, double& tmp_dx,
	double& tmp_dy, double& tmp_dz, long long& tmp_loc, long long* table, hiprandState* states, int index, long long& vox_idx)
{
	bool reject = false; //reject move or not
	//first, identify which voxel the particle is in
	long long x_pos = floor(tmp_x / w->vox_size);
	long long y_pos = floor(tmp_y / w->vox_size);
	long long z_pos = floor(tmp_z / w->vox_size);
	double p_ex, p_12, p_21;
	long long xyz, old_vox_idx; //voxel identifier
	old_vox_idx = vox_idx; //save curent identifier in case particle is rejected after move

	pair(x_pos, y_pos, z_pos, xyz); //get the identifier

	vox_idx = binary_search_iter(table, 0, w->num_voxels - 1, xyz); //iterative binary search

	if (vox_idx != -1) //"now intra"
	{
		if (tmp_loc == 1) { reject = false; } //was intra before
		else //was not intra before
		{
			//compute permeation probability p_21
			p_ex = (double)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0)); //from Szafer 1995 and others, for any geoemetry
			p_12 = (double)p_ex * (1 - w->f1);
			p_21 = (double)p_ex * w->f1;
			if (hiprand_uniform_double(&states[index]) < p_21) { reject = false; tmp_loc = 1; }
			else { reject = true; tmp_loc = 0; }
		}
	}

	if (vox_idx == -1) //"now extra"
	{
		if (tmp_loc == 0) { reject = false; } //was extra before
		else //was not extra before
		{
			//compute permeation probability p_12
			p_ex = (double)opt->kappa * sqrt(8 * opt->sim_dt / (3 * opt->D0)); //from Szafer 1995 and others, for any geoemetry
			p_12 = (double)p_ex * (1 - w->f1);
			p_21 = (double)p_ex * w->f1;
			if (hiprand_uniform(&states[index]) < p_12) { reject = false; tmp_loc = 0; }
			else { reject = true; tmp_loc = 1; }
		}
	}


	if (reject) { tmp_x -= tmp_dx; tmp_y -= tmp_dy; tmp_z -= tmp_dz; vox_idx = old_vox_idx; }
	//restore voxel index if step is rejected.
}

__global__ //the global keyword tells compiler this is device code not host code
void engine(double* x, double* y, double* z, long long* loc, long long* table,
	hiprandState* states, options* opt, world* w, double* phase, double * gwf_x, double *gwf_y, double* gwf_z)
{
	long index = blockIdx.x * blockDim.x + threadIdx.x;
	//int stride = blockDim.x * gridDim.x;

	long entry, phase_entry;
	double tmp_x, tmp_y, tmp_z;
	double tmp_dx, tmp_dy, tmp_dz;
	double e_x, e_y, e_z; //keep track of hyperposition
	long long tmp_loc;
	long long vox_idx; // voxel index, needed by check_state

	//for (int c_p = index; c_p < (*opt).Npart; c_p += stride)
	long c_p = index;
	if (c_p < opt->Npart) //ensure we keep within bounds
	{
		tmp_x = x[c_p];
		tmp_y = y[c_p];
		tmp_z = z[c_p];
		tmp_dx = 0;
		tmp_dy = 0;
		tmp_dz = 0;
		tmp_loc = loc[c_p];

		//each thread needs to know the initial voxel id of its particle
		long long start_x_pos = floor(tmp_x / w->vox_size);
		long long start_y_pos = floor(tmp_y / w->vox_size);
		long long start_z_pos = floor(tmp_z / w->vox_size);
		long long start_xyz; //voxel identifier
		pair(start_x_pos, start_y_pos, start_z_pos, start_xyz); //get the identifier
		vox_idx = binary_search_iter(table, 0, w->num_voxels - 1, start_xyz); //iterative binary search

		//printf("vox_idx : %lld\n", vox_idx);
		
		e_x = 0; e_y = 0; e_z = 0;
		//adding a delay loop here
		
		for (int c_t = 0; c_t < (*opt).sim_Nt; c_t++)
		{
			move(tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, entry, states, index, opt);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
			check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
				tmp_dy, tmp_dz, tmp_loc, table, states, index, vox_idx);
			restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
		}

		e_x = 0; e_y = 0; e_z = 0;

		//do signal-related calculations here
			  
			   //run the simulation
			for (int c_t = 0; c_t < (*opt).sim_Nt; c_t++)
				{
					move(tmp_x, tmp_y, tmp_z, tmp_dx,
					tmp_dy, tmp_dz, entry, states, index, opt);
					restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);
					check_state(w, opt, tmp_x, tmp_y, tmp_z, tmp_dx,
					tmp_dy, tmp_dz, tmp_loc, table, states, index, vox_idx);
					restrict_to_world(e_x, e_y, e_z, opt, w, tmp_x, tmp_y, tmp_z);

				for (int c_a = 0; c_a < opt->n_acq; c_a++)
					{
						 phase_entry = c_p * (*opt).n_acq + c_a;
			//printf("phase_entry : %f\n", tmp_y);
				phase[phase_entry] += opt->gamma*((tmp_x+e_x)*gwf_x[c_a*(*opt).sim_Nt+c_t] + (tmp_y+e_y)*gwf_y[c_a*(*opt).sim_Nt+c_t] + (tmp_z+e_z)*gwf_z[c_a*(*opt).sim_Nt+c_t])*opt->sim_dt;
				//if (c_p==0){printf("phase_entry : %f\n", gwf_x[c_a*(*opt).sim_Nt+c_t]);}
				//if (c_p==2){printf("phase_entry : %ld\n", phase_entry);}
				//if (c_p==2 && c_a==1){printf("phase_entry : %f\n", gwf_x[c_a*(*opt).sim_Nt+c_t]);}
					}
			}
	}
}



void set_options(options* opt, char* pos_fn, char* sig_fn, char* sub_fn, char* gwf_fn, char* sta_fn)
{
	//sets options loaded from file
	std::string opt_fn = "C:\\Users\\Arthur\\source\\repos\\PaSiD\\pasid_random_signal_opt.txt";
	std::string dummy;

	std::ifstream tf;
	tf.open(opt_fn, std::ios::in);
	tf >> dummy >> opt->Npart;
	tf >> dummy >> opt->sim_dt;
	tf >> dummy >> opt->samp_dt;
	tf >> dummy >> opt->D0;
	tf >> dummy >> opt->kappa;
	tf >> dummy >> sub_fn;
	tf >> dummy >> gwf_fn;
	tf >> dummy >> sig_fn;
	tf >> dummy >> opt->save_positions;
	tf >> dummy >> pos_fn;
	tf >> dummy >> opt->save_states;
	tf >> dummy >> sta_fn;
	tf.close();

	opt->delay = 500; //how many steps to take before starting to acquire signal
	opt->gamma = 2.675129e8;
	opt->n_dim = 3;
	opt->ds = (double)sqrt(2 * (*opt).n_dim * (*opt).D0 * (*opt).sim_dt); //step size
	
	std::cout << "Loaded options from: " << opt_fn << std::endl;
}


void save_trajectory(double* x, double* y, double* z, char* pos_fn, options* opt)
{
	std::cout << "Saving trajectory to: " << pos_fn << std::endl;
	FILE* tf;
	tf = fopen(pos_fn, "wb");
	fwrite(&(opt->Npart), sizeof(long long), 1, tf);
	fwrite(&(opt->T), sizeof(double), 1, tf);
	fwrite(&(opt->save_Nt), sizeof(long long), 1, tf);
	fwrite(x, sizeof(double), opt->N_save, tf); // opt->N_save
	fwrite(y, sizeof(double), opt->N_save, tf);
	fwrite(z, sizeof(double), opt->N_save, tf);
	fclose(tf);
	std::cout << "Done." << std::endl;
}


void save_state_history(long long* s, char* sta_fn, options* opt)
{
	//saves history of particle identities/compartment identities
	std::cout << "Saving state history to: " << sta_fn << std::endl;
	FILE* tf;
	tf = fopen(sta_fn, "wb");
	fwrite(&(opt->Npart), sizeof(long long), 1, tf);
	fwrite(&(opt->T), sizeof(double), 1, tf);
	fwrite(&(opt->save_Nt), sizeof(long long), 1, tf);
	fwrite(s, sizeof(long long), opt->Npart, tf); // opt->N_save
	fclose(tf);
	std::cout << "Done." << std::endl;
}


void save_signal(double* h_signal, char* sig_fn, options* opt)
{
	
	std::cout << "### Checking contents of signal..." << h_signal[0] << " " << h_signal[8] << std::endl;
	//saves signals to file
	std::cout << "Saving signals to: " << sig_fn << std::endl;
	FILE* tf;
	tf = fopen(sig_fn, "wb");
	fwrite(h_signal, sizeof(double), opt->n_acq, tf);
	fclose(tf);
	std::cout << "Done." << std::endl;
}

void get_num_voxels(world* w, char* sub_fn)
{
	//open substrate file and get number of voxels
	FILE* sf;
	sf = fopen(sub_fn, "rb");
	fread(&(w->num_voxels), sizeof(long long), 1, sf);
	fclose(sf);
}


void get_gwf_info(char* gwf_fn, options* opt)
{
	std::cout << "Getting gwf info from: " << gwf_fn << std::endl;
	//open gwf file and get number of b_values and number of time points in gwf
	FILE* sf;
	sf = fopen(gwf_fn, "rb");
	fread(&(opt->n_acq), sizeof(long long), 1, sf);
	fread(&(opt->n_gwf_points), sizeof(long long), 1, sf);
	fclose(sf);
	
	//total simulation time is number of points in gwf*sim_dt
	opt->sim_Nt = opt->n_gwf_points;
	//set this also in variable T
	opt->T = (opt->sim_Nt-1)*opt->sim_dt;
	
	opt->save_Nt = (long long)round(opt->T / opt->samp_dt);
	opt->N_save = (long long)opt->Npart * opt->save_Nt; //N time points x N particles
	opt->N_sim = (long long)opt->Npart * opt->sim_Nt;
	
	std::cout << "Done." << std::endl;
}


void load_substrate(long long* h_table, world* w, char* g_fn)
{
	//load substrate from file
	FILE* sf;
	sf = fopen(g_fn, "rb");
	fread(&(w->num_voxels), sizeof(long long), 1, sf);
	fread(&(w->vox_size), sizeof(double), 1, sf);
	fread(&(w->max_x), sizeof(double), 1, sf);
	fread(&(w->max_y), sizeof(double), 1, sf);
	fread(&(w->max_z), sizeof(double), 1, sf);
	fread(h_table, sizeof(long long), w->num_voxels, sf); //now expecting user to supply lookup table
	fread(&w->f1, sizeof(double), 1, sf);
	fclose(sf);

	w->x_length = 2 * w->max_x;
	w->y_length = 2 * w->max_y;
	w->z_length = 2 * w->max_z;

	std::cout << "Loaded substrate from: " << g_fn << std::endl;
}


void load_gwf(char* gwf_fn, options* opt, double* h_gwf_x, double* h_gwf_y, double* h_gwf_z)
{
	std::cout << "Loading gradient waveform..."<< std::endl;
	//load substrate from file
	FILE* sf;
	sf = fopen(gwf_fn, "rb");
	fread(&(opt->n_acq), sizeof(long long), 1, sf);
	fread(&(opt->n_gwf_points), sizeof(long long), 1, sf);
	fread(h_gwf_x, sizeof(double), opt->n_acq*opt->n_gwf_points, sf);
	fread(h_gwf_y, sizeof(double), opt->n_acq*opt->n_gwf_points, sf);
	fread(h_gwf_z, sizeof(double), opt->n_acq*opt->n_gwf_points, sf); 
	fclose(sf);

	std::cout << "$$$ Checking gwf: point 3 " << h_gwf_x[2] << " point 11: "<< h_gwf_x[10] <<std::endl;
	std::cout << "Loaded gradient waveform from: " << gwf_fn << std::endl;
}


void convert_phase_to_signal(double *h_phase, double *h_signal, options *opt)
{
	//convert phase to signal
	std::cout << "Converting phase to signal..." << std::endl;
	double sum_cos_phase;
	long long phase_entry;
	for (int ca = 0; ca<opt->n_acq; ca++)
	{
		sum_cos_phase = 0;
	for (long c_p = 0; c_p < opt->Npart; c_p++)
		{
			phase_entry = c_p * (*opt).n_acq + ca;
			sum_cos_phase += cos(h_phase[phase_entry]);
		}
		h_signal[ca] = sum_cos_phase/opt->Npart;
	}
	/*/
	for (int ca = 0; ca < opt->n_acq; ca++)
	{
		sum_cos_phase = 0;
		for (int cp = ca; cp < opt->n_acq*opt->Npart; cp+=opt->n_acq)
		{
			sum_cos_phase += cos(h_phase[cp]);
		}
		h_signal[ca] = sum_cos_phase/opt->Npart;
	 }/*/
	 std::cout << "Done." << std::endl;
}


void save_phase(double* h_phase, options *opt, char* sta_fn)
{
		//saves phase to file
		long long N = opt->Npart*opt->n_acq;
	std::cout << "Saving phase to: " << sta_fn << std::endl;
	FILE* tf;
	tf = fopen(sta_fn, "wb");
	fwrite(&N, sizeof(long long), 1, tf);
	fwrite(h_phase, sizeof(double), opt->n_acq*opt->Npart, tf);
	fclose(tf);
	std::cout << "Done." << std::endl;
}




__global__ void generate_initial_distribution(double* x, double* y, double* z, long long* loc, long long* table,
	hiprandState* states, options* opt, world* w)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x; //get thread idx
	double tmp_x, tmp_y, tmp_z, frac = 0.5;
	long long tmp_xll, tmp_yll, tmp_zll, tmp_xyz, tmp_vox_idx;
	long long idx;
	bool success_1 = false, success_2 = false, success = false;
	if (w->f1 == 0) { success_2 = true; frac = 1; if( index == 0) {printf("Detected f1 = 0.");};}
	int N1 = (int)(frac * opt->Npart);
	if (index < N1)
	{
		//places particles in initial positions all over substrate

		while (!success)
		{
			tmp_x = -w->max_x + 2 * hiprand_uniform_double(&states[index]) * w->max_x; //suggest initial position
			tmp_y = -w->max_y + 2 * hiprand_uniform_double(&states[index]) * w->max_y; //suggest initial position
			tmp_z = -w->max_z + 2 * hiprand_uniform_double(&states[index]) * w->max_z; //suggest initial position
			tmp_xll = (long long)floor(tmp_x / w->vox_size);
			tmp_yll = (long long)floor(tmp_y / w->vox_size);
			tmp_zll = (long long)floor(tmp_z / w->vox_size);
			pair(tmp_xll, tmp_yll, tmp_zll, tmp_xyz); //get pair
			tmp_vox_idx = binary_search_iter(table, 0, w->num_voxels - 1, tmp_xyz); //check what this position corresponds to

			//printf("%lld \n", tmp_vox_idx);
			
			if (tmp_vox_idx != -1 && !success_1)
			{
				idx = index;
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 1;
				success_1 = true;
			}


			if (tmp_vox_idx == -1 && !success_2)
			{
				idx = (N1 + index);
				x[idx] = tmp_x;
				y[idx] = tmp_y;
				z[idx] = tmp_z;
				loc[idx] = 0;
				success_2 = true;
			}
			success = success_1 && success_2;
		}
	}
}


void generate_lookup_table(world* h_w, long long* h_world_x, long long* h_world_y, long long* h_world_z, long long* h_table)
{
	//build lookup table using the szudzik pairing algorithm
	std::cout << "Building voxel lookup table..." << std::endl;
	long long x, y, z, xy, xyz;
	for (int c = 0; c < h_w->num_voxels; c++)
	{
		x = h_world_x[c];
		y = h_world_y[c];
		z = h_world_z[c];

		x >= 0 ? x = 2 * x : x = -2 * x - 1;
		y >= 0 ? y = 2 * y : y = -2 * y - 1;
		z >= 0 ? z = 2 * z : z = -2 * z - 1;

		x >= y ? xy = x * x + x + y : xy = y * y + x;
		xy >= 0 ? xy = 2 * xy : xy = -2 * xy - 1;

		xy >= z ? xyz = xy * xy + xy + z : xyz = z * z + xy;

		h_table[c] = xyz;
	}
	std::cout << "Done." << std::endl;
}


int main(void)
{

	std::clock_t start;
	double duration;
	start = std::clock();

	hipError_t error = hipSuccess;

	int nDevices;
	hipGetDeviceCount(&nDevices);
	printf("Number of devices: %d\n", nDevices);
	int activeDevice;
	hipGetDevice(&activeDevice);
	printf("Active device index: %d\n", activeDevice);
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, activeDevice);
	printf("Device name: %s\n", prop.name);


	//non-numerical options, host-only
	char pos_fn[200], sig_fn[200], sub_fn[200], gwf_fn[200], sta_fn[200]; //filenames for final positions, signal,  substrate, waveform, states

	//world structure for device and host
	world* h_w, * dev_w;
	h_w = (world*)malloc(sizeof(world));
	hipMalloc(&dev_w, sizeof(world));


	//options structure for host
	options* opt, * dev_opt;
	opt = (options*)malloc(sizeof(options));
	set_options(opt, pos_fn, sig_fn, sub_fn, gwf_fn, sta_fn);
	
	//--GRADIENT WAVEFORM--
	//__________________________________________________________________
	double *h_gwf_x, *h_gwf_y, *h_gwf_z; //waveform in x y z
	double *dev_gwf_x, *dev_gwf_y, *dev_gwf_z;
	//get number of b-values and number of time points in gwf
	
	get_gwf_info(gwf_fn, opt); //need this to allocate memory for arrays above. Note: this function updates opt
	
	std::cout << "Number of acquisitions: " << opt->n_acq << std::endl;
	std::cout << "Number of time points per waveform: " << opt->n_gwf_points << std::endl;
	
	//allocate on host
	h_gwf_x = (double*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(double));
	h_gwf_y = (double*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(double));
	h_gwf_z = (double*)malloc(opt->n_acq*opt->n_gwf_points* sizeof(double));
	//allocate on device
	hipMalloc(&dev_gwf_x, opt->n_acq*opt->n_gwf_points * sizeof(double));
	hipMalloc(&dev_gwf_y, opt->n_acq*opt->n_gwf_points * sizeof(double));
	hipMalloc(&dev_gwf_z, opt->n_acq*opt->n_gwf_points * sizeof(double));
	//load the waveform and b-values
	load_gwf(gwf_fn, opt, h_gwf_x, h_gwf_y, h_gwf_z);
	//copy to device
	hipMemcpy(dev_gwf_x, h_gwf_x, opt->n_acq*opt->n_gwf_points * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_gwf_y, h_gwf_y, opt->n_acq*opt->n_gwf_points * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_gwf_z, h_gwf_z, opt->n_acq*opt->n_gwf_points * sizeof(double), hipMemcpyHostToDevice);
	//__________________________________________________________________		
	hipMalloc(&dev_opt, sizeof(options));
	hipMemcpy(dev_opt, opt, sizeof(options), hipMemcpyHostToDevice);

	//we will load the simulation world from file, no need to waste time implementing it in here
	//the first entry in the substrate file will be the number of voxels in the world
	//this is so we know how large the world arrays world_x, world_y, world_z and mask need to be

	get_num_voxels(h_w, sub_fn);
	std::cout << "Num vox intra: " << h_w->num_voxels << std::endl;
	//now we declare world arrays on device and host
	long long* h_table;
	long long* dev_table;

	//allocate on host
	h_table = (long long*)malloc(h_w->num_voxels * sizeof(long long));
	//allocate on device
	hipMalloc(&dev_table, h_w->num_voxels * sizeof(long long));
	//load the substrate
	load_substrate(h_table, h_w, sub_fn);

	std::cout << "Num voxels: " << h_w->num_voxels << " Vox size: " << h_w->vox_size << " max_z: " << h_w->max_z << std::endl;

	//copy substrate data to GPU
	hipMemcpy(dev_w, h_w, sizeof(world), hipMemcpyHostToDevice);

	std::cout << " Example of table entry at ten: " << h_table[9] << std::endl;

	//copy the lookup table to the GPU
	error = hipMemcpy(dev_table, h_table, h_w->num_voxels * sizeof(long long), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cout << "FAILED TO ALLOCATE SUBSTRATE MEMORY ON GPU." << std::endl;
		//throw error;
	}


	//--SIGNAL--
	//declare signal and phase arrays
	double* h_signal, *h_phase; //for the host
	double * dev_phase; //for the device
	//allocate them on host
	h_signal = (double*)malloc(opt->n_acq * sizeof(double));
	h_phase = (double*)malloc(opt->n_acq*opt->Npart * sizeof(double));
	//initialise phase array, it's important that it contains only zeros
	for (int c = 0; c < opt->n_acq*opt->Npart; c++) {h_phase[c] = 0;}
	//allocate memory for arrays on device
	hipMalloc(&dev_phase, opt->n_acq*opt->Npart * sizeof(double));
	//copy phase array to device
	hipMemcpy(dev_phase, h_phase, opt->n_acq*opt->Npart * sizeof(double), hipMemcpyHostToDevice);
	//__________________________________________________________________	

	//declare traj arrays and particle location (compartment id)
	long long* h_loc, * dev_loc;
	double* h_x, * h_y, * h_z; //for the host
	double* dev_x, * dev_y, * dev_z; //for the device
	//allocate them on host
	h_loc = (long long*)malloc(opt->Npart* sizeof(long long));
	h_x = (double*)malloc(opt->Npart * sizeof(double));
	h_y = (double*)malloc(opt->Npart * sizeof(double));
	h_z = (double*)malloc(opt->Npart * sizeof(double));


	//allocate memory for arrays on device
	hipMalloc(&dev_loc, opt->Npart * sizeof(long long));
	hipMalloc(&dev_x, opt->Npart * sizeof(double));
	hipMalloc(&dev_y, opt->Npart * sizeof(double));
	hipMalloc(&dev_z, opt->Npart * sizeof(double));

	//copy x,y,z  and id arrays to device
	hipMemcpy(dev_loc, h_loc, opt->Npart * sizeof(long long), hipMemcpyHostToDevice);
	hipMemcpy(dev_x, h_x, opt->Npart * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_y, h_y, opt->Npart * sizeof(double), hipMemcpyHostToDevice);
	error = hipMemcpy(dev_z, h_z, opt->Npart * sizeof(double), hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cout << "FAILED TO ALLOCATE TRAJECTORY MEMORY ON GPU." << std::endl;
		//throw error;
	}
	else { std::cout << "Generating initial distribution..." << std::endl; }

	//define grid texture
	int blockSize;
	512 > opt->Npart ? blockSize = (int)opt->Npart : blockSize = 512;
	int numBlocks = (int)(opt->Npart + blockSize - 1) / blockSize; //make sure to round up in case N is not an integer multiple of blockSize

	//allcoate hiprandState for every CUDA thread on the host
	hiprandState* dev_states;
	hipMalloc(&dev_states, blockSize * numBlocks * sizeof(hiprandState));
	//initialise RNG for all threads
	random_init << < numBlocks, blockSize >> > (dev_states);
	//generate initial particle distribution
	generate_initial_distribution << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_table, dev_states, dev_opt, dev_w);
	//launch simulation engine
	std::cout << "Running simulation..." << std::endl;
	engine << < numBlocks, blockSize >> > (dev_x, dev_y, dev_z, dev_loc, dev_table, dev_states, dev_opt, dev_w, 
	dev_phase, dev_gwf_x, dev_gwf_y, dev_gwf_z);

	hipDeviceSynchronize(); //Tell CPU to wait until kernel is done before accessing results. This is necessary because
							//cuda kernel launches do not block the calling CPU thread.

	std::cout << "Simulation complete. Downloading results..." << std::endl;

	//--DOWNLOAD RESULTS--
	//copy phase array back to host machine
	hipMemcpy(h_phase, dev_phase, opt->n_acq*opt->Npart * sizeof(double), hipMemcpyDeviceToHost);
	//copy final particle positions for diagnostics
	hipMemcpy(h_x, dev_x, opt->Npart * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_y, dev_y, opt->Npart * sizeof(double), hipMemcpyDeviceToHost);
	hipMemcpy(h_z, dev_z, opt->Npart * sizeof(double), hipMemcpyDeviceToHost);
	//__________________________________________________________________

	std::cout << "Done." << std::endl;
	
	convert_phase_to_signal(h_phase, h_signal, opt);
	
	save_phase(h_phase, opt, sta_fn);
	
	save_signal(h_signal, sig_fn, opt);
	
	//write results to binary files
	if (opt->save_positions) {save_trajectory(h_x, h_y, h_z, pos_fn, opt);};
	if (opt->save_states) {save_state_history(h_loc, sta_fn, opt);};

	duration = (std::clock() - start) / (double)CLOCKS_PER_SEC;
	std::cout << "Elapsed time is: " << duration << " seconds." << std::endl;


	// Free memory on host
	free(h_signal);
	free(h_phase);
	free(h_gwf_x);
	free(h_gwf_y);
	free(h_gwf_z);
	free(h_x);
	free(h_y);
	free(h_z);
	free(h_w);
	free(opt);
	free(h_loc);
	free(h_table);
	//free memory on device
	//free memory on device
	hipFree(dev_phase);
	hipFree(dev_gwf_x);
	hipFree(dev_gwf_y);
	hipFree(dev_gwf_z);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(dev_states);
	hipFree(dev_loc);
	hipFree(dev_w);
	hipFree(dev_opt);
	hipFree(dev_table);
	return 0;
}
